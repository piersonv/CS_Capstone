#include "hip/hip_runtime.h"
// by Olaf Hall-Holt, 2007-2015
#include <iostream>
#include <string>
#include "eriolHeader.h"
#include "homography.h"
#include "NCCDemo.h"
#include "time.h"
#include <cstdlib>
#include <math.h>
#include "book.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

__device__ double dev_correlation;

__global__ void calculate_correlation_CUDA(const thrust::device_vector<Color> & first_signal, const thrust::device_vector<Color> & second_signal, double * signal_correlationR_thread, double * signal_correlationG_thread, double * signal_correlationB_thread)
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int size_of_signal = first_signal.size();

  double signal_correlationR = 0;
  double signal_correlationG = 0;
  double signal_correlationB = 0;

  signal_correlationR_thread = first_signal[tid].r*second_signal[tid].r;
  signal_correlationG_thread = first_signal[tid].g*second_signal[tid].g;
  signal_correlationB_thread = first_signal[tid].b*second_signal[ctid].b;

  if(tid == 0){
    for (int i = 0; i < size_of_signal; i++){
      signal_correlationR += signal_correlationR_thread[i];
      signal_correlationG += signal_correlationG_thread[i];
      signal_correlationB += signal_correlationB_thread[i];
    }
    dev_correlation = (signal_correlationR+signal_correlationG+signal_correlationB)/3
  }
}

double calculate_normalized_correlation_CUDA(const thrust::host_vector<Color> & first_signal, const thrust::host_vector<Color> & second_signal)
{
  int size_of_signal = first_signal.size();
  
  //CUDA Variables
  double * signal_correlationR_thread;
  double * signal_correlationG_thread;
  double * signal_correlationB_thread;
  Color * dev_first_signal = first_signal;
  Color * dev_second_signal = second_signal;

  //CUDA Allocation
  hipMalloc(&signal_correlationR_thread, size_of_signal*sizeof(double));
  hipMalloc(&signal_correlationG_thread, size_of_signal*sizeof(double));
  hipMalloc(&signal_correlationB_thread, size_of_signal*sizeof(double));
  hipMalloc(&dev_first_signal, size_of_signal*sizeof(Color));
  hipMalloc(&dev_second_signal, size_of_signal*sizeof(Color));

  //Calculate block and thread size
  double block_num = round(sqrt(size_of_signal));
  double thread_num = size_of_signal/block_num;

  double correlation; 
  calculate_correlation_CUDA<<<block_num,thread_num>>>(dev_first_signal, dev_second_signal, signal_correlationR, signal_correlationG, signal_correlationB);
  hipMemcpy(&correlation, dev_correlation, sizeof(double), hipMemcpyDeviceToHost);

  double sum_first_signalR = 0;
  double sum_second_signalR = 0;
  double sum_first_signalG = 0;
  double sum_second_signalG = 0;
  double sum_first_signalB = 0;
  double sum_second_signalB = 0;
  int correlation_scalar = 0;

  for (int count = 0; count < size_of_signal; count++)
  {
    sum_first_signalR += first_signal[count].r * first_signal[count].r;
    sum_second_signalR += second_signal[count].r * second_signal[count].r;
    sum_first_signalG += first_signal[count].g * first_signal[count].g;
    sum_second_signalG += second_signal[count].g * second_signal[count].g;
    sum_first_signalB += first_signal[count].b * first_signal[count].b;
    sum_second_signalB += second_signal[count].b * second_signal[count].b;
  }
  double total_sum1 = (sum_first_signalR + sum_first_signalG + sum_first_signalB) / 3; 
  double total_sum2 = (sum_second_signalR + sum_second_signalG + sum_second_signalB) / 3; 
  correlation_scalar = sqrt(total_sum1*total_sum2);

  return (double)correlation/correlation_scalar;

}

double calcNCC(vector<PixelLoc> *interior, double * current, Image *myimg, Image *myimgOther)
{
	Color black(0,0,0);
 	Color white(255,255,255);
	thrust::host_vector<Color> signal1,signal2;
	double point[2];
	for(unsigned int i=0; i<interior->size(); ++i){
		homography(interior[0][i].x + 0.5 , interior[0][i].y + 0.5, current, point);
                Coord mycoord(point[0], point[1]);
                if(inImage(myimg,mycoord)){
                    signal1.push_back(asInterpolatedColor(mycoord, myimg));
                } else {
                    signal1.push_back(black);
                    signal2.push_back(white);
                    continue;
                }
                signal2.push_back(myimgOther->getPixel(interior[0][i]));
         }
         return calculate_normalized_correlation_CUDA(signal1, signal2);
}


int main(int argc, char **argv)
{
  cout << "Starting" << endl;
  double point[2];
  double * current = new double[9];
  double * best = new double[9];
  double * init = new double[9];
  float ncc;
  float bestncc = -2;
  float first;
  double scale = 0.1;
  bool initial = true;
  //int position = 0;
  //int direction = 1;
  bool optimize = true;

 cout << "Starting" << endl; 
 vector<PixelLoc> interior;
 for(int i=5; i<=10; ++i){
	for(int j=5; j<=10; ++j){
	PixelLoc point(i, j);
	interior.push_back(point);
	}
 }
cout << "Creating Images" << endl;
 Image myimg("test-initial.ppm");
 Image myimgOther("test-final.ppm");
cout << "Images created" << endl;
 for(int i=0;i<9;++i){
		init[i] = current[i] = best[i] = 0; 
 }
  	init[8] = current[8] = best[8] = 1;
  	init[0] = current[0] = best[0] = 1;
  	init[4] = current[4] = best[4] = 1;
  

Color red(255,0,0);
Color blue(0,0,100);

Image imgInitial = myimg;
Image src = myimgOther;

for(unsigned int i=0; i<interior.size(); ++i){
   homography(interior[i].x, interior[i].y, current, point);
   PixelLoc loc((int)point[0], (int)point[1]);
   if(inImage(&imgInitial,loc)){
       imgInitial.setPixel(loc,blue);
   }
}

imgInitial.print("initial.ppm");

for(unsigned int i=0; i<interior.size(); ++i){
   if(inImage(&src,interior[i])){
      src.setPixel(interior[i],blue);
   }
}
src.print("src.ppm");

int count;
cout << endl;
if(optimize){
	for(double i=1;i<=1000000;i*=10){
	count = 0;
 	cout <<"Scale = " << (scale/i) << endl;
		
	double offset = -50*(scale/i);
	for(int l=0; l<2; ++l){
	for(int k=0; k < 8; ++k){
		for(int j=1; j<=100; ++j){
 			ncc = calcNCC(&interior, current, &myimg, &myimgOther);
			if (initial){
				first = ncc;
   				initial = false;
			}
			if (ncc > bestncc){
				l=0;
	   	       		++count;
				bestncc = ncc;
				best[k] = current[k];
			}
     		randHomography(k, init, current, offset + (scale/i)*j);
    		}
		cout << "Count: " << count << endl;
		count = 0;
		for(int j=0; j<9; ++j){
			init[j] = current[j] =  best[j];
		}	
	}
	}
	
	}
}
 
 cout << "First: " << first << " Best: " << bestncc << endl;
 cout << "homography: "; 
 for(int i=0;i<9;++i){
	cout << current[i] << " ";
 } 
 cout << endl;
Image imgFinal = myimg;

for(unsigned int i=0; i<interior.size(); ++i){
   homography(interior[i].x, interior[i].y, best, point);
   PixelLoc loc((int)point[0], (int)point[1]);
   if(inImage(&imgInitial,loc)){
      imgFinal.setPixel(loc,blue);
   }
}
imgFinal.print("final.ppm");
system("/home/mscs/bin/show src.ppm initial.ppm final.ppm");
}
