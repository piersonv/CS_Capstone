#include "hip/hip_runtime.h"
// by Olaf Hall-Holt, 2007-2015
#include <iostream>
#include <string>
#include "time.h"
#include "glareReduction.h" 
#include <cstdlib>
#include <math.h>
#include "book.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

Tile &loadJustOneTile(const string &tileID, const string &imgName);
vector<PixelLoc> getPixelsFor(int);

__device__ double dev_correlation;

__global__ void calculate_correlation_CUDA(const thrust::device_vector<Color> & first_signal, const thrust::device_vector<Color> & second_signal, double * signal_correlationR_thread, double * signal_correlationG_thread, double * signal_correlationB_thread);
{
  unsigned int tid = threadIdx.x + blockIdx.x * numBlock;
  int size_of_signal = first_signal.size();

  double signal_correlationR = 0;
  double signal_correlationG = 0;
  double signal_correlationB = 0;

  signal_correlationR_thread = first_signal[tid].r*second_signal[tid].r;
  signal_correlationG_thread = first_signal[tid].g*second_signal[tid].g;
  signal_correlationB_thread = first_signal[tid].b*second_signal[ctid].b;

  if(tid == 0){
    for (int i = 0; i < size_of_signal; i++){
      signal_correlationR += signal_correlationR_thread[i];
      signal_correlationG += signal_correlationG_thread[i];
      signal_correlationB += signal_correlationB_thread[i];
    }
    dev_correlation = (signal_correlationR+signal_correlationG+signal_correlationB)/3
  }
}

double calculate_normalized_correlation_CUDA(const thrust::host_vector<Color> & first_signal, const thrust::host_vector<Color> & second_signal)
{
  int size_of_signal = first_signal.size();
  
  //CUDA Variables
  double * signal_correlationR_thread;
  double * signal_correlationG_thread;
  double * signal_correlationB_thread;
  thrust::device_vector<Color> dev_first_signal = first_signal;
  thrust::device_vector<Color> dev_second_signal = second_signal;

  //CUDA Allocation
  hipMalloc(&signal_correlationR_thread, size_of_signal*sizeof(double));
  hipMalloc(&signal_correlationG_thread, size_of_signal*sizeof(double));
  hipMalloc(&signal_correlationB_thread, size_of_signal*sizeof(double));

  //Calculate block and thread size
  double block_num = round(sqrt(size_of_signal));
  double thread_num = size_of_signal/block_num;

  double correlation; 
  calculate_correlation_CUDA<<<block_num,thread_num>>>(dev_first_signal, dev_second_signal, signal_correlationR, signal_correlationG, signal_correlationB);
  hipMemcpy(&correlation, dev_correlation, sizeof(double), hipMemcpyDeviceToHost);

  double sum_first_signalR = 0;
  double sum_second_signalR = 0;
  double sum_first_signalG = 0;
  double sum_second_signalG = 0;
  double sum_first_signalB = 0;
  double sum_second_signalB = 0;
  int correlation_scalar = 0;

  for (int count = 0; count < size_of_signal; count++)
  {
    sum_first_signalR += first_signal[count].r * first_signal[count].r;
    sum_second_signalR += second_signal[count].r * second_signal[count].r;
    sum_first_signalG += first_signal[count].g * first_signal[count].g;
    sum_second_signalG += second_signal[count].g * second_signal[count].g;
    sum_first_signalB += first_signal[count].b * first_signal[count].b;
    sum_second_signalB += second_signal[count].b * second_signal[count].b;
  }
  double total_sum1 = (sum_first_signalR + sum_first_signalG + sum_first_signalB) / 3; 
  double total_sum2 = (sum_second_signalR + sum_second_signalG + sum_second_signalB) / 3; 
  correlation_scalar = sqrt(total_sum1*total_sum2);

  return (double)correlation/correlation_scalar;

}

double calcNCC(vector<PixelLoc> *interior, double * current, Image *myimg, Image *myimgOther)
{
  int 
  Color black(0,0,0);
  Color white(255,255,255);
  thrust::host_vector<Color> signal1, signal2;
  double point[2];
  for(unsigned int i=0; i<interior->size(); ++i){
    homography(interior[0][i].x + 0.5 , interior[0][i].y + 0.5, current, point);
                Coord mycoord(point[0], point[1]);
                if(inImage(myimg,mycoord)){
                    signal1.push_back(asInterpolatedColor(mycoord, myimg));
                } else {
                    signal1.push_back(black);
                    signal2.push_back(white);
                    continue;
                }
                signal2.push_back(myimgOther->getPixel(interior[0][i]));
         }
         return calculate_normalized_correlation_CUDA(signal1, signal2);
}


int main(int argc, char **argv)
{
  //CPU variables
  string tile = argv[1];
  string image = argv[2];
  string imageR = image+"R";
  string imageL = image+"L";
  double point[2];
  double * current = new double[9];
  double * best = new double[9];
  double * init = new double[9];
  float ncc;
  float bestncc = -2;
  float first;
  double scale = 0.01;
  bool initial = true;
  int position = 0;
  int direction = 1;
  bool optimize = true;

  vector<PixelLoc> interiorR = getContour(tile, imageR);
  vector<PixelLoc> interiorL = getContour(tile, imageL);
  vector<Coord> fpSource;
  vector<Coord> fpDestination;
  vector<Coord> fpR = getFeaturePoints(tile, imageR);
  vector<Coord> fpL = getFeaturePoints(tile, imageL);
   Matrix3x3 myH1;
   vector<PixelLoc> interior;
   Image myimg;
   Image myimgOther;

  if (interiorR.size() > interiorL.size())
  {
    interior = getContour(tile, imageL);
    myH1  = getHomography(tile, imageL, imageR);
    myimg = imageR.c_str();
    myimgOther = imageL.c_str();
    fpSource = getFeaturePoints(tile, imageL); 
    fpDestination = getFeaturePoints(tile, imageR); 
  }
  else
  {
    interior = getContour(tile, imageR);
    myH1  = getHomography(tile, imageR, imageL);
    myimg = imageL.c_str();
    myimgOther = imageR.c_str();
    fpSource = getFeaturePoints(tile, imageR);
    fpDestination = getFeaturePoints(tile, imageL);

  }
  if(argc == 12){
  optimize = false;
  for(int i=0;i<9;++i){
          current[i] = best[i] = (double)atof(argv[3+i]);
  }
    current[8] = best[8] = 1; 
  }else if(argc == 21){
  optimize = false;
  cout << "Comparing" << endl;
        for(int i=0;i<18;++i){
                if (i < 9){
      current[i] = (double)atof(argv[3+i]);
    } else {
      best[i-9] = (double)atof(argv[3+i]);
    } 
        }
        current[8] = best[8] = 1;
  }else{
    cerr << "homography: " << myH1 << endl; 
    for(int i=0;i<9;++i){
    init[i] = current[i] = best[i] = myH1.m[i];
    }
    init[8] = current[8] = best[8] = 1;
  }
  cout << "Original Feature points: " ;
  for(unsigned int i=0;i<fpDestination.size();++i){
        cout << fpDestination[i] << " ";
  }

Color red(255,0,0);
Color blue(0,0,100);

Image imgInitial = myimg;
Image src = myimgOther;
for(unsigned int i=0; i<interior.size(); ++i){
   homography(interior[i].x, interior[i].y, current, point);
   PixelLoc loc((int)point[0], (int)point[1]);
   if(inImage(&imgInitial,loc)){
       imgInitial.setPixel(loc,blue);
   }
}
for(unsigned int i=0;i<fpDestination.size();++i){
   homography(fpSource[i].x,fpSource[i].y, current, point);
   PixelLoc loc((int)point[0], (int)point[1]);
   if(inImage(&imgInitial,loc)){
        imgInitial.setPixel(loc,red);
   }
}
imgInitial.print("initial.ppm");
for(unsigned int i=0; i<interior.size(); ++i){
   if(inImage(&src,interior[i])){
      src.setPixel(interior[i],blue);
   }
}
for(unsigned int i=0;i<fpDestination.size();++i){
   PixelLoc loc((int)fpSource[i].x, (int)fpSource[i].y);
   if(inImage(&src,loc)){ 
       src.setPixel(loc,red);
   }
}
src.print("src.ppm");
int count;
cout << endl;
if(optimize){
  for(double i=1;i<=1000000;i*=10){
  count = 0;
  cout <<"Scale = " << (scale/i) << endl;
    
  double offset = -50*(scale/i);
  for(int l=0; l<2; ++l){
  for(int k=0; k < 8; ++k){
    for(int j=1; j<=100; ++j){
      ncc = calcNCC(&interior, current, &myimg, &myimgOther);
      if (initial){
        first = ncc;
          initial = false;
      }
      if (ncc > bestncc){
        l=0;
                ++count;
        bestncc = ncc;
        best[k] = current[k];
      }
        randHomography(k, init, current, offset + (scale/i)*j);
        }
    cout << "Count: " << count << endl;
    count = 0;
    for(int j=0; j<9; ++j){
      init[j] = current[j] =  best[j];
    } 
  }
  }
  
  }
}
// cout << "Best so far: " << bestncc << endl;
// cout << "homography: ";
// for(int i=0;i<9;++i){
//        cout << best[i] << " ";
// }
// cout << endl;

Image imageWithGlare = myimg;


 int ncc_glare_reduced = calculateNCCWithoutGlare(&interior, best, &myimg, &myimgOther, &imageWithGlare);
 cout << "First: " << first << " Best: " << bestncc <<  "Glare Reduced: " << ncc_glare_reduced << endl;
 cout << "homography: "; 
 for(int i=0;i<9;++i){
  cout << current[i] << " ";
 } 
 cout << endl;
// cout << "Origonal Feature points: " ;
// for(unsigned int i=0;i<fpDestination.size();++i){
//  cout << fpDestination[i] << " ";
// }
 cout << endl << "Optimized Feature points: ";
 for(unsigned int i=0;i<fpDestination.size();++i){
      homography(fpSource[i].x,fpSource[i].y, best, point);
      cout << point[0] << "," << point[1] << " ";
 } 
 cout << endl;
Image imgFinal = myimg;

for(unsigned int i=0; i<interior.size(); ++i){
   homography(interior[i].x, interior[i].y, best, point);
   PixelLoc loc((int)point[0], (int)point[1]);
   if(inImage(&imgInitial,loc)){
      imgFinal.setPixel(loc,blue);
   }
}
for(unsigned int i=0;i<fpDestination.size();++i){
   homography(fpSource[i].x,fpSource[i].y, best, point);
   PixelLoc loc((int)point[0], (int)point[1]);
   if(inImage(&imgInitial,loc)){
      imgFinal.setPixel(loc,red);
   }
}
imgFinal.print("final.ppm");
imageWithGlare.print("glare.ppm");
system("/home/mscs/bin/show src.ppm initial.ppm final.ppm glare.ppm");
}
