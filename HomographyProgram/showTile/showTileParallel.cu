// by Olaf Hall-Holt, 2007-2015
#include <iostream>
#include <string>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "eriolHeader.h"
#include "../homography.h"
#include "NCCDemo.h"
#include "time.h"

Tile &loadJustOneTile(const string &tileID, const string &imgName);
vector<PixelLoc> getPixelsFor(int);

//Global variables
string tile;
string image;
string imageR;
string imageL;
double * best = new double[9];
float bestncc = -2;
float first;
double scale = 1;
bool initial = true;
thrust::host_vector<PixelLoc> interiorR;
thrust::host_vector<PixelLoc> interiorL;
thrust::host_vector<PixelLoc> interior;
Image myimg;
Image myimgOther;

void runNCC(double * current, int j, int k){
  //main variables
  double point[2];
  thrust::device_vector<Color> d_intcolors = intcolors;
  thrust::device_vector<Color> d_intcolors2 = intcolors2;
  float ncc;
  for(unsigned int i=0; i<interior.size(); ++i){
  homography(interior[i].x + 0.5 , interior[i].y + 0.5, current, point);
    Coord mycoord(point[0], point[1]);
    if (point[0] < myimg.getWidth() && point[1] < myimg.getHeight()){
              intcolors.push_back(asInterpolatedColor(mycoord, &myimg));
    } else {
      continue;
    }
    intcolors2.push_back(myimgOther.getPixel(interior[i]));
   }
   ncc = calculate_normalized_correlation(intcolors, intcolors2);
   if (initial){
    first = ncc;
      initial = false;
    cout << "Initial: " << first << endl;
   }

   if (ncc > bestncc){
    bestncc = ncc;
    for(int i=0;i<9;++i){
      best[i] = current[i];
    }
    j=0;
   }
   long seed = (long)time(NULL) * j;
   //cout << seed << endl;
         randHomography(best, current, seed, scale/k);
   intcolors.clear();
   intcolors2.clear();
}

int main(int argc, char **argv)
{
  //main variables
  // double point[2];
  // /*thrust::host_*/vector<Color> intcolors;
  // /*thrust::host_*/vector<Color> intcolors2;
  double * current = new double[9];
  // float ncc;
  Matrix3x3 myH1;

  // //GPU variables
  // double * d_point;
  // float d_ncc; //in function
  // Matrix3x3 d_myH1;

  tile = argv[1];
  image = argv[2];
  imageR = image+"R";
  imageL = image+"L";
  interiorR = getContour(tile, imageR);
  interiorL = getContour(tile, imageL);

  best = new double[9];

  if (interiorR.size() > interiorL.size())
  {
    interior = getContour(tile, imageL);
    myH1  = getHomography(tile, imageL, imageR);
    myimg = imageR.c_str();
    myimgOther = imageL.c_str();
  }
  else
  {
    interior = getContour(tile, imageR);
    myH1  = getHomography(tile, imageR, imageL);
    myimg = imageL.c_str();
    myimgOther = imageR.c_str();
  }

  cerr << "homography: " << myH1 << endl; 
  for(int i=0;i<9;++i){
  current[i] = myH1.m[i];
  } 

  // //CUDA allocation
  // hipMalloc(d_point, 2*sizeof(double));
  // hipMalloc(d_current, 9*sizeof(double));

  // //CUDA copying
  

  for(int k=10; k < 100000; k*=10){
  cout << "Trying with scale = " << scale/k << endl;

  float d_bestncc = -2;
  double * d_best;
    

  runNCC(current, j, k);


 cout << "Best so far: " << bestncc << endl;
 cout << "homography: ";
 for(int i=0;i<9;++i){
        cout << best[i] << " ";
 }
 cout << endl;
 }
 cout << "First: " << first << " Best: " << bestncc << endl;
 cout << "homography: "; 
 for(int i=0;i<9;++i){
  cout << best[i] << " ";
 } 
 cout << endl;
 
}
